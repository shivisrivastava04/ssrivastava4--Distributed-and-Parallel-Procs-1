
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define N 6000  /* Matrix size */

float A[N][N], B[N][N];

int threadPerBlock=256;
int block=(int)N/256;

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row][col] = (float)rand() / 32768.0;
            B[row][col] = 0.0;
        }
    }
    
}
//Kernel Processing

__global__ void matrixNorm(float *d_A, float *d_B, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; //unique id of thread within the grid
    int row;
    float mu, sigma;
    if (col < n){
        mu = (float)0.0;
        for (row=0; row < n; row++)
            mu += d_A[col*n+row];
        mu /= (float) n;

        __syncthreads();

        sigma = (float)0.0;
        for (row=0; row < n; row++)
            sigma += powf(d_A[col*n+row] - mu, (float)2.0);
        sigma /= (float) n;

        __syncthreads();

        sigma = sqrt(sigma);

        for (row=0; row < n; row++) {
            if (sigma == (float)0.0)
                d_B[row*n+col] = (float)0.0;
            else
                d_B[row*n+col] = (d_A[col*n+row] - mu) / sigma;
        }
    }
}








int  main(int argc, char **argv) {

float *d_A, *d_B;
if(hipMalloc((void **) &d_A, sizeof(float)*N*N)!=hipSuccess){
return 0;
}
if(hipMalloc((void **) &d_B, sizeof(float)*N*N)!=hipSuccess){
hipFree(d_A);
return 0;
}

if(hipMemcpy(d_A, A, sizeof(float)*N*N, hipMemcpyHostToDevice)!=hipSuccess){
hipFree(d_A);
hipFree(d_B);
return 0;
}


    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;
    
    /* Initialize A and B */
    initialize_inputs();
    
    
    /* Start Clock */
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    gettimeofday(&start, &tzdummy);
    
    
    /* Matrix Normalization */
    matrixNorm<<<block,threadPerBlock>>>(d_A,d_B,N);
 
    if(hipMemcpy(B,d_B, sizeof(float)*N*N,hipMemcpyDeviceToHost)!=hipSuccess){
hipFree(d_A);
hipFree(d_B);
return 0;
}   
else{
hipFree(d_A);
hipFree(d_B);
}
    
    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);
    
    
    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");
    
    exit(0);
}
